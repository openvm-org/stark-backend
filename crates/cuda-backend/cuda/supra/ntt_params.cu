#include "hip/hip_runtime.h"
/*
 * Source: https://github.com/supranational/sppark (tag=v0.1.12)
 * Status: MODIFIED from sppark/ntt/parameters.cuh
 * Imported: 2025-08-13 by @gaxiom
 * 
 * LOCAL CHANGES (high level):
 * - 2025-08-13: NTTParameters constructor async on custom stream
 * - 2025-08-26: NTTParameters constructor on hipStreamPerThread
 * - 2025-09-05: Stop using __constant__ for twiddles[0]
 * - 2025-09-10: Delete NTTParameters & add extern "C" launcher
 * - 2025-10-02: move all twiddles to __constant__
 */

#include "launcher.cuh"
#include "ntt/parameters.cuh"

__constant__ fr_t FORWARD_TWIDDLES[TWIDDLES_SIZE];
__constant__ fr_t INVERSE_TWIDDLES[TWIDDLES_SIZE];
__constant__ fr_t FORWARD_PARTIAL_TWIDDLES[WINDOW_NUM][WINDOW_SIZE];
__constant__ fr_t INVERSE_PARTIAL_TWIDDLES[WINDOW_NUM][WINDOW_SIZE];

__global__ void generate_all_twiddles(fr_t* d_radixX_twiddles, 
    const fr_t root6, const fr_t root7, const fr_t root8, const fr_t root9, const fr_t root10)
{
    const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int pow = 0;
    fr_t root_of_unity;

    if (tid < 32) {
        pow = tid;
        root_of_unity = root6;
    } else if (tid < 32 + 64) {
        pow = tid - 32;
        root_of_unity = root7;
    } else if (tid < 32 + 64 + 128) {
        pow = tid - 32 - 64;
        root_of_unity = root8;
    } else if (tid < 32 + 64 + 128 + 256) {
        pow = tid - 32 - 64 - 128;
        root_of_unity = root9;
    } else if (tid < 32 + 64 + 128 + 256 + 512) {
        pow = tid - 32 - 64 - 128 - 256;
        root_of_unity = root10;
    } else {
        assert(false);
    }

    d_radixX_twiddles[tid] = root_of_unity^pow;
}

__global__ void generate_partial_twiddles(fr_t (*roots)[WINDOW_SIZE],
                               const fr_t root_of_unity)
{
    const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    assert(tid < WINDOW_SIZE);
    fr_t root;

    root = root_of_unity^tid;

    roots[0][tid] = root;

    for (int off = 1; off < WINDOW_NUM; off++) {
        for (int i = 0; i < LG_WINDOW_SIZE; i++)
            root.sqr();
        roots[off][tid] = root;
    }
}

extern "C" int _generate_all_twiddles(fr_t* twiddles, bool inverse) {
    const fr_t* roots = inverse ? inverse_roots_of_unity : forward_roots_of_unity;

    generate_all_twiddles<<<TWIDDLES_SIZE/32, 32>>>(
            twiddles, roots[6], roots[7], roots[8], roots[9], roots[10]);

    if (inverse) {
        hipMemcpyToSymbolAsync(HIP_SYMBOL(INVERSE_TWIDDLES), twiddles, TWIDDLES_SIZE * sizeof(fr_t),
                                0, hipMemcpyDeviceToDevice, hipStreamPerThread);
    } else {
        hipMemcpyToSymbolAsync(HIP_SYMBOL(FORWARD_TWIDDLES), twiddles, TWIDDLES_SIZE * sizeof(fr_t),
                                0, hipMemcpyDeviceToDevice, hipStreamPerThread);
    }
    hipStreamSynchronize(hipStreamPerThread);
    return CHECK_KERNEL();
}

extern "C" int _generate_partial_twiddles(fr_t (*partial_twiddles)[WINDOW_SIZE], bool inverse) {
    const fr_t* roots = inverse ? inverse_roots_of_unity : forward_roots_of_unity;
    generate_partial_twiddles<<<WINDOW_SIZE/32, 32>>>(
            partial_twiddles, roots[MAX_LG_DOMAIN_SIZE]);

    if (inverse) {
        hipMemcpyToSymbolAsync(HIP_SYMBOL(INVERSE_PARTIAL_TWIDDLES), partial_twiddles, WINDOW_NUM * WINDOW_SIZE * sizeof(fr_t),
                                0, hipMemcpyDeviceToDevice, hipStreamPerThread);
    } else {
        hipMemcpyToSymbolAsync(HIP_SYMBOL(FORWARD_PARTIAL_TWIDDLES), partial_twiddles, WINDOW_NUM * WINDOW_SIZE * sizeof(fr_t),
                                0, hipMemcpyDeviceToDevice, hipStreamPerThread);
    }
    hipStreamSynchronize(hipStreamPerThread);
    return CHECK_KERNEL();
}