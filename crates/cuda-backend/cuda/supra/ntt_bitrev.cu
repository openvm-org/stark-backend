#include "hip/hip_runtime.h"
/*
 * Source: https://github.com/supranational/sppark (tag=v0.1.12)
 * Status: MODIFIED from sppark/ntt/kernels.cu
 * Imported: 2025-08-13 by @gaxiom
 * 
 * LOCAL CHANGES (high level):
 * - 2025-08-13: Support multiple rows in bit_rev_permutation & bit_rev_permutation_z
 * - 2025-09-10: Add extern "C" launcher from sppark/ntt/ntt.cuh
 */

#include <cstdint>

#include "launcher.cuh"
#include "ntt/ntt.cuh"

namespace {
uint32_t max_grid_dim_y() {
    int device = 0;
    if (hipGetDevice(&device) != hipSuccess)
        return 65535u;

    int attr = 0;
    if (hipDeviceGetAttribute(&attr, hipDeviceAttributeMaxGridDimY, device) != hipSuccess)
        return 65535u;

    return attr > 0 ? static_cast<uint32_t>(attr) : 65535u;
}
} // namespace

// Permutes the data in an array such that data[i] = data[bit_reverse(i)]
// and data[bit_reverse(i)] = data[i]
__launch_bounds__(1024) __global__
void bit_rev_permutation(fr_t* d_out, const fr_t *d_in, uint32_t lg_domain_size,
                         uint32_t padded_poly_size, uint32_t poly_count)
{
    const uint32_t poly_idx = blockIdx.y + blockIdx.z * gridDim.y;
    if (poly_idx >= poly_count)
        return;
    d_out += static_cast<size_t>(poly_idx) * padded_poly_size; // [DIFF]: move out ptr to another row
    d_in += static_cast<size_t>(poly_idx) * padded_poly_size;  // [DIFF]: move in ptr to another row

    if (gridDim.x == 1 && blockDim.x == (1 << lg_domain_size)) {
        uint32_t idx = threadIdx.x;
        uint32_t rev = bit_rev(idx, lg_domain_size);

        fr_t t = d_in[idx];
        if (d_out == d_in)
            __syncthreads();
        d_out[rev] = t;
    } else {
        index_t idx = threadIdx.x + blockDim.x * (index_t)blockIdx.x;
        index_t rev = bit_rev(idx, lg_domain_size);
        bool copy = d_out != d_in && idx == rev;

        if (idx < rev || copy) {
            fr_t t0 = d_in[idx];
            if (!copy) {
                fr_t t1 = d_in[rev];
                d_out[idx] = t1;
            }
            d_out[rev] = t0;
        }
    }
}

template<unsigned int Z_COUNT>
__launch_bounds__(192, 2) __global__
void bit_rev_permutation_z(fr_t* out, const fr_t* in, uint32_t lg_domain_size,
                           uint32_t padded_poly_size, uint32_t poly_count)
{
    const uint32_t poly_idx = blockIdx.y + blockIdx.z * gridDim.y;
    if (poly_idx >= poly_count)
        return;
    out += static_cast<size_t>(poly_idx) * padded_poly_size;   // [DIFF]: move out ptr to another row
    in += static_cast<size_t>(poly_idx) * padded_poly_size;    // [DIFF]: move in ptr to another row

    const uint32_t LG_Z_COUNT = 31 - __clz(Z_COUNT); // [DIFF]: use __clz to get lg2

    extern __shared__ fr_t xchg[][Z_COUNT][Z_COUNT];

    uint32_t gid = threadIdx.x / Z_COUNT;
    uint32_t idx = threadIdx.x % Z_COUNT;
    uint32_t rev = bit_rev(idx, LG_Z_COUNT);

    index_t step = (index_t)1 << (lg_domain_size - LG_Z_COUNT);
    index_t tid = threadIdx.x + blockDim.x * (index_t)blockIdx.x;

    #pragma unroll 1
    do {
        index_t group_idx = tid >> LG_Z_COUNT;
        index_t group_rev = bit_rev(group_idx, lg_domain_size - 2*LG_Z_COUNT);

        if (group_idx > group_rev)
            continue;

        index_t base_idx = group_idx * Z_COUNT + idx;
        index_t base_rev = group_rev * Z_COUNT + idx;

        fr_t regs[Z_COUNT];

        #pragma unroll
        for (uint32_t i = 0; i < Z_COUNT; i++) {
            xchg[gid][i][rev] = (regs[i] = in[i * step + base_idx]);
            if (group_idx != group_rev)
                regs[i] = in[i * step + base_rev];
        }

        (Z_COUNT > WARP_SIZE) ? __syncthreads() : __syncwarp();

        #pragma unroll
        for (uint32_t i = 0; i < Z_COUNT; i++)
            out[i * step + base_rev] = xchg[gid][rev][i];

        if (group_idx == group_rev)
            continue;

        (Z_COUNT > WARP_SIZE) ? __syncthreads() : __syncwarp();

        #pragma unroll
        for (uint32_t i = 0; i < Z_COUNT; i++)
            xchg[gid][i][rev] = regs[i];

        (Z_COUNT > WARP_SIZE) ? __syncthreads() : __syncwarp();

        #pragma unroll
        for (uint32_t i = 0; i < Z_COUNT; i++)
            out[i * step + base_idx] = xchg[gid][rev][i];

    } while (Z_COUNT <= WARP_SIZE && (tid += blockDim.x*gridDim.x) < step);
    // without "Z_COUNT <= WARP_SIZE" compiler spills 128 bytes to stack
}


extern "C" int _bit_rev(fr_t* d_out, const fr_t* d_inp, 
    uint32_t lg_domain_size, uint32_t padded_poly_size, uint32_t poly_count)
{
    assert(lg_domain_size <= MAX_LG_DOMAIN_SIZE);

    size_t domain_size = (size_t)1 << lg_domain_size;
    // aim to read 4 cache lines of consecutive data per read
    const uint32_t Z_COUNT = 256 / sizeof(fr_t);
    const uint32_t bsize = Z_COUNT > WARP_SIZE ? Z_COUNT : WARP_SIZE;

    if (poly_count == 0)
        return hipSuccess;

    const uint32_t max_y = max_grid_dim_y();
    const uint64_t total_polys = poly_count;
    const uint64_t max_y_64 = max_y == 0 ? 1 : static_cast<uint64_t>(max_y);
    uint32_t grid_z = static_cast<uint32_t>((total_polys + max_y_64 - 1) / max_y_64);
    if (grid_z == 0)
        grid_z = 1;
    uint64_t grid_y_64 = (total_polys + grid_z - 1) / grid_z;
    uint32_t grid_y = static_cast<uint32_t>(grid_y_64);
    if (grid_y > max_y)
        grid_y = max_y == 0 ? 1 : max_y;

    // [DIFF]: N -> dim3(N, poly_count) in grid_size; stream -> hipStreamPerThread
    if (domain_size <= 1024)
        bit_rev_permutation<<<dim3(1u, grid_y, grid_z), domain_size>>>
                            (d_out, d_inp, lg_domain_size, padded_poly_size, poly_count);
    else if (domain_size < bsize * Z_COUNT)
        bit_rev_permutation<<<dim3(static_cast<unsigned int>(domain_size / WARP_SIZE), grid_y, grid_z), WARP_SIZE>>>
                            (d_out, d_inp, lg_domain_size, padded_poly_size, poly_count);
    else if (Z_COUNT > WARP_SIZE || lg_domain_size <= 32)
        bit_rev_permutation_z<Z_COUNT><<<dim3(static_cast<unsigned int>(domain_size / Z_COUNT / bsize), grid_y, grid_z), bsize,
                                            bsize * Z_COUNT * sizeof(fr_t)>>>
                            (d_out, d_inp, lg_domain_size, padded_poly_size, poly_count);
    else {
        // Those GPUs that can reserve 96KB of shared memory can
        // schedule 2 blocks to each SM...
        int device;
        hipGetDevice(&device);
        int sm_count;
        hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device);

        bit_rev_permutation_z<Z_COUNT><<<dim3(static_cast<unsigned int>(sm_count * 2), grid_y, grid_z), 192,
                                            192 * Z_COUNT * sizeof(fr_t)>>>
                                (d_out, d_inp, lg_domain_size, padded_poly_size, poly_count);
    }

    return CHECK_KERNEL();
}
