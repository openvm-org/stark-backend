#include "hip/hip_runtime.h"
/*
 * Source: https://github.com/scroll-tech/plonky3-gpu (private repo)
 * Status: BASED ON plonky3-gpu/gpu-backend/src/cuda/kernels/matrix.cu
 * Imported: 2025-01-25 by @gaxiom
 */

#include "fp.h"
#include "fpext.h"
#include "launcher.cuh"

const size_t TILE_SIZE = 32; // do not change,
// const uint64_t GROUP_SIZE = 32; // do not change,

template <typename T>
__global__ void __launch_bounds__(TILE_SIZE)
    cukernel_matrix_transpose(T *output, const T *input, size_t col_size, size_t row_size) {
#ifndef __clang_analyzer__
    // NOTE: This is what builds, we need to use this for clang-tidy to work
    __shared__ T s_mem[TILE_SIZE][TILE_SIZE + 1];
#else
    __shared__ __align__(alignof(T)
    ) unsigned char s_mem_raw[TILE_SIZE * (TILE_SIZE + 1) * sizeof(T)];
    T(*s_mem)[TILE_SIZE + 1] = reinterpret_cast<T(*)[TILE_SIZE + 1]>(s_mem_raw);
#endif
    size_t dim_x = (col_size + TILE_SIZE - 1) / TILE_SIZE;
    size_t bid = blockIdx.x; // (x, 1, 1)
    size_t bid_y = bid / dim_x;
    size_t bid_x = bid % dim_x; // (bid_x, bid_y, 1)

    size_t tid = threadIdx.x;
    size_t index_i = bid_y * TILE_SIZE * col_size + bid_x * TILE_SIZE + tid;
    size_t index_o = bid_x * TILE_SIZE * row_size + bid_y * TILE_SIZE + tid;

    // input
    bool boundray_column = bid_x * TILE_SIZE + tid < col_size;
    size_t row_offset = bid_y * TILE_SIZE + 0;
    for (auto i = 0; i < TILE_SIZE; ++i) {
        bool boundray = boundray_column && (row_offset + i < row_size);
        s_mem[i][tid] = (boundray) ? input[index_i + i * col_size] : T(0);
    }
    __syncthreads();

    // output
    boundray_column = bid_y * TILE_SIZE + tid < row_size;
    row_offset = bid_x * TILE_SIZE + 0;
    for (auto i = 0; i < TILE_SIZE; ++i) {
        bool boundray = boundray_column && (row_offset + i < col_size);
        if (boundray)
            output[index_o + i * row_size] = s_mem[tid][i];
    }
}

// Explicit instantiations
template __global__ void cukernel_matrix_transpose<Fp>(Fp *, const Fp *, size_t, size_t);
template __global__ void cukernel_matrix_transpose<FpExt>(FpExt *, const FpExt *, size_t, size_t);

__global__ void cukernel_matrix_get_rows_fp(
    Fp *output,
    const Fp *input,
    uint32_t *row_indices,
    uint64_t matrix_width,
    uint64_t matrix_height
) {
    uint32_t col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (col_idx >= matrix_width) {
        return;
    }
    // Note: blockIdx.y may >= matrix_height, do not assert

    uint64_t input_row = row_indices[blockIdx.y];
    uint64_t output_row = blockIdx.y;
    uint64_t input_idx = col_idx * matrix_height + input_row;  // col-major matrix
    uint64_t output_idx = output_row * matrix_width + col_idx; // row-major matrix
    output[output_idx] = input[input_idx];
}

// plonky3/matrix/src/lib.rs: fn vertically_strided
__global__ void cukernel_split_ext_poly_to_multiple_base_matrix(
    const uint64_t *d_matrix_ptr, // array of matrices over Fp
    FpExt *d_poly,
    uint64_t poly_len,
    uint64_t num_chunk
) {
    uint64_t row_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (row_idx >= poly_len) { // [0, poly_len - 1]
        return;
    }

    uint64_t chunk_idx = row_idx % num_chunk;
    Fp *d_matrix = (Fp *)d_matrix_ptr[chunk_idx];
    // assumption:
    //   1. FpExt::D = 4, the input FpExt will be flatten to Fp
    //   2. the order of the matrix is column major
    // uint64_t matrix_width = 4; // FpExt::D
    uint64_t matrix_height = (poly_len / num_chunk);
    uint64_t remainder = poly_len % num_chunk;
    if (chunk_idx < remainder) {
        matrix_height += 1;
    }
    uint64_t chunk_row = row_idx / num_chunk;

    FpExt ext_val = d_poly[row_idx];
    d_matrix[matrix_height * 0 + chunk_row] = ext_val.elems[0];
    d_matrix[matrix_height * 1 + chunk_row] = ext_val.elems[1];
    d_matrix[matrix_height * 2 + chunk_row] = ext_val.elems[2];
    d_matrix[matrix_height * 3 + chunk_row] = ext_val.elems[3];
}

// LAUNCHERS

template <typename T>
int matrix_transpose_impl(T *output, const T *input, size_t col_size, size_t row_size) {
    uint32_t grid_x = (col_size + TILE_SIZE - 1) / TILE_SIZE;
    uint32_t grid_y = (row_size + TILE_SIZE - 1) / TILE_SIZE;

    dim3 grid(grid_x * grid_y);
    dim3 block(TILE_SIZE);

    cukernel_matrix_transpose<T><<<grid, block>>>(output, input, col_size, row_size);

    return hipGetLastError();
}

extern "C" int matrix_transpose_fp(Fp *output, const Fp *input, size_t col_size, size_t row_size) {
    return matrix_transpose_impl(output, input, col_size, row_size);
}

extern "C" int matrix_transpose_fpext(
    FpExt *output,
    const FpExt *input,
    size_t col_size,
    size_t row_size
) {
    return matrix_transpose_impl(output, input, col_size, row_size);
}

extern "C" int _split_ext_poly_to_multiple_base_matrix(
    const uint64_t *d_matrix_ptr,
    FpExt *d_poly,
    uint64_t poly_len,
    uint64_t num_chunk
) {
    auto [grid, block] = kernel_launch_params(poly_len);
    cukernel_split_ext_poly_to_multiple_base_matrix<<<grid, block>>>(
        d_matrix_ptr, d_poly, poly_len, num_chunk
    );
    return hipGetLastError();
}

extern "C" int _matrix_get_rows_fp(
    Fp *output,
    const Fp *input,
    uint32_t *row_indices,
    uint64_t matrix_width,
    uint64_t matrix_height,
    uint32_t row_indices_len
) {
    auto block = WARP_SIZE;
    dim3 grid = dim3(div_ceil(matrix_width, WARP_SIZE), row_indices_len);
    cukernel_matrix_get_rows_fp<<<grid, block>>>(
        output, input, row_indices, matrix_width, matrix_height
    );
    return hipGetLastError();
}
