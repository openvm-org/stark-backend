#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>

extern "C" {

// Return 0 on success, else CUresult / cudaError_t code

int _vpmm_check_support(int device_ordinal) {
    hipDevice_t dev;
    hipError_t r = hipDeviceGet(&dev, device_ordinal);
    if (r != hipSuccess) return (int)r;
    int vpmm = 0;
    r = hipDeviceGetAttribute(&vpmm, hipDeviceAttributeVirtualMemoryManagementSupported, dev);
    if (r != hipSuccess) return (int)r;
    return vpmm ? 0 : (int)hipErrorNotSupported;
}

int _vpmm_min_granularity(int device_ordinal, size_t* out) {
    if (!out) return (int)hipErrorInvalidValue;
    hipDevice_t dev;
    hipError_t r = hipDeviceGet(&dev, device_ordinal);
    if (r != hipSuccess) return (int)r;

    hipMemAllocationProp prop{};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device_ordinal;
    prop.requestedHandleTypes = hipMemHandleTypeNone;

    return (int)hipMemGetAllocationGranularity(out, &prop, hipMemAllocationGranularityMinimum);
}

int _vpmm_reserve(size_t size, size_t align, hipDeviceptr_t* out_va) {
    if (!out_va) return (int)hipErrorInvalidValue;
    return (int)hipMemAddressReserve(out_va, size, align, 0, 0);
}

int _vpmm_release_va(hipDeviceptr_t base, size_t size) {
    return (int)hipMemAddressFree(base, size);
}

int _vpmm_create_physical(int device_ordinal, size_t bytes, hipMemGenericAllocationHandle_t* out_h) {
    if (!out_h) return (int)hipErrorInvalidValue;
    hipMemAllocationProp prop{};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device_ordinal;
    prop.requestedHandleTypes = hipMemHandleTypeNone;
    return (int)hipMemCreate(out_h, bytes, &prop, 0);
}

int _vpmm_map(hipDeviceptr_t va, size_t bytes, hipMemGenericAllocationHandle_t h) {
    return (int)hipMemMap(va, bytes, 0, h, 0);
}

int _vpmm_set_access(hipDeviceptr_t va, size_t bytes, const int device_ordinal) {
    const hipMemAccessDesc acc = {
        .location = {
            .type = hipMemLocationTypeDevice,
            .id = device_ordinal
        },
        .flags = hipMemAccessFlagsProtReadWrite
    };
    return (int)hipMemSetAccess(va, bytes, &acc, 1);
}

int _vpmm_unmap(hipDeviceptr_t va, size_t bytes) {
    return (int)hipMemUnmap(va, bytes);
}

int _vpmm_release(hipMemGenericAllocationHandle_t h) {
    return (int)hipMemRelease(h);
}

} // extern "C"
