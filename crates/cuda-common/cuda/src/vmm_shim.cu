#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>

extern "C" {

// Return 0 on success, else CUresult / cudaError_t code

int _vmm_check_support(int device_ordinal) {
  hipDevice_t dev;
  hipError_t r = hipDeviceGet(&dev, device_ordinal);
  if (r != hipSuccess) return (int)r;
  int vmm = 0;
  r = hipDeviceGetAttribute(&vmm, hipDeviceAttributeVirtualMemoryManagementSupported, dev);
  if (r != hipSuccess) return (int)r;
  return vmm ? 0 : (int)hipErrorNotSupported;
}

int _vmm_min_granularity(int device_ordinal, size_t* out) {
  if (!out) return (int)hipErrorInvalidValue;
  hipDevice_t dev;
  hipError_t r = hipDeviceGet(&dev, device_ordinal);
  if (r != hipSuccess) return (int)r;

  hipMemAllocationProp prop{};
  prop.type = hipMemAllocationTypePinned;
  prop.location.type = hipMemLocationTypeDevice;
  prop.location.id = device_ordinal;
  prop.requestedHandleTypes = hipMemHandleTypeNone;

  return (int)hipMemGetAllocationGranularity(out, &prop, hipMemAllocationGranularityRecommended);
}

int _vmm_reserve(size_t size, size_t align, hipDeviceptr_t* out_va) {
  if (!out_va) return (int)hipErrorInvalidValue;
  return (int)hipMemAddressReserve(out_va, size, align, 0, 0);
}

int _vmm_release_va(hipDeviceptr_t base, size_t size) {
  return (int)hipMemAddressFree(base, size);
}

int _vmm_create_physical(int device_ordinal, size_t bytes, hipMemGenericAllocationHandle_t* out_h) {
  if (!out_h) return (int)hipErrorInvalidValue;
  hipMemAllocationProp prop{};
  prop.type = hipMemAllocationTypePinned;
  prop.location.type = hipMemLocationTypeDevice;
  prop.location.id = device_ordinal;
  prop.requestedHandleTypes = hipMemHandleTypeNone;
  // leave win32HandleMetaData/reserved/allocFlags as zero-init
  return (int)hipMemCreate(out_h, bytes, &prop, 0);
}

int _vmm_map_and_set_access(hipDeviceptr_t va, size_t bytes, hipMemGenericAllocationHandle_t h, int device_ordinal) {
  hipError_t r = hipMemMap(va, bytes, 0, h, 0);
  if (r != hipSuccess) return (int)r;
  hipMemAccessDesc acc{};
  acc.location.type = hipMemLocationTypeDevice;
  acc.location.id = device_ordinal;
  acc.flags = hipMemAccessFlagsProtReadWrite;
  return (int)hipMemSetAccess(va, bytes, &acc, 1);
}

int _vmm_unmap_release(hipDeviceptr_t va, size_t bytes, hipMemGenericAllocationHandle_t h) {
  hipError_t r = hipMemUnmap(va, bytes);
  if (r != hipSuccess) return (int)r;
  return (int)hipMemRelease(h);
}

} // extern "C"
