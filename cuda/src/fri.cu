#include "hip/hip_runtime.h"
// FROM https://github.com/scroll-tech/plonky3-gpu/blob/openvm-v2/gpu-backend/src/cuda/kernels/fri.cu

#include "fpext.h"
#include "launcher.cuh"

const uint32_t TILE_WIDTH = 32;

__forceinline__ __device__ uint32_t bit_rev(uint32_t x, uint32_t n) {
    return __brev(x) >> (__clz(n) + 1);
}

// result[i] = (1/2 + beta/2 g_inv^i) * folded[2*i]
//           + (1/2 - beta/2 g_inv^i) * folded[2*i+1]
//           + fri_input[i]
__global__ void cukernel_fri_fold(
    FpExt *result,
    FpExt *folded,
    const FpExt *fri_input,
    FpExt *d_constants,
    Fp *g_inv_powers,
    uint64_t N
) {
    FpExt half_beta = d_constants[0]; // beta/2
    FpExt half_one = d_constants[1];  // 1/2
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        FpExt beta_g_inv = half_beta * g_inv_powers[idx]; // beta/2 * g_inv^i
        FpExt c1 = half_one + beta_g_inv;                 // 1/2 + beta/2 * g_inv^i
        FpExt c2 = half_one - beta_g_inv;                 // 1/2 - beta/2 * g_inv^i
        FpExt a = folded[2 * idx];
        FpExt b = folded[2 * idx + 1];
        FpExt res = c1 * a;
        res += c2 * b;
        if (fri_input != nullptr) {
            res += fri_input[idx];
        }
        result[idx] = res;
    }
}

// compute diffs = { (z - shift*g^j) } for j in 0..N
__global__ void compute_diffs(FpExt *diffs, FpExt *d_z, Fp *d_domain, uint32_t log_n) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;

    Fp shift = d_domain[0];
    Fp g = d_domain[1];
    uint32_t N = 1 << log_n;
    Fp g_idx = pow(g, idx);
    Fp g_pow = pow(g, stride);
    FpExt z = *d_z;

    for (; idx < N; idx += stride, g_idx *= g_pow) {
        FpExt diff = z - shift * g_idx;
        diffs[idx] = diff;
    }
}

// data[i] = g^i for i in 0..N
__global__ void powers(Fp *data, Fp *d_g, uint32_t N) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;
    Fp g = *d_g;
    Fp g_idx = pow(g, idx);
    Fp g_pow = pow(g, stride);

    for (; idx < N; idx += stride, g_idx *= g_pow) {
        data[idx] = g_idx;
    }
}

__global__ void powers_ext(FpExt *data, FpExt *d_g, uint32_t N) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;
    FpExt g = *d_g;
    FpExt g_idx = pow(g, idx);
    FpExt g_pow = pow(g, stride);

    for (; idx < N; idx += stride, g_idx *= g_pow) {
        data[idx] = g_idx;
    }
}

__global__ void precompute_diff_powers(
    FpExt *d_output,
    const FpExt *diff_invs,
    const Fp *powers,
    uint32_t N
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;
    for (; idx < N; idx += stride) {
        if (idx < N) {
            d_output[idx] = diff_invs[idx] * powers[idx];
        }
    }
}

__global__ void fri_bit_reverse(FpExt *data, uint32_t log_n) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;
    uint32_t N = 1 << log_n;
    for (; idx < N; idx += stride) {
        uint32_t ridx = bit_rev(idx, N);
        if (idx < ridx) {
            FpExt tmp = data[idx];
            data[idx] = data[ridx];
            data[ridx] = tmp;
        }
    }
}

// batch inversion
__global__ void batch_invert(FpExt *data, uint64_t log_n) {

    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;
    const uint32_t batch_size = 16;

    uint32_t N = 1ULL << log_n;
    if (idx >= N) {
        return;
    }

    // a, ab, abc, abcd, ...
    FpExt accums[batch_size];
    accums[0] = data[idx];

    uint32_t j = 1;
    uint32_t pos = idx + stride;
    for (; (j < batch_size) && pos < N; pos += stride, j += 1) {
        accums[j] = accums[j - 1] * data[pos];
    }

    j -= 1;
    pos -= stride;
    // accum_inv = inv(prod(data[idx], data[idx+stride], ...,
    // data[idx+(j-1)*stride]))
    FpExt accum_inv = binomial_inversion(accums[j]);

    for (; j > 0; pos -= stride, j -= 1) {
        FpExt tmp = accum_inv * accums[j - 1]; // inv(data[pos])
        accum_inv *= data[pos];
        data[pos] = tmp;
    }

    data[idx] = accum_inv;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
/// recall that barycentric algorithm for getting the evaluation of polynomial
/// p(x) at a random point z given its evaluations over coset domain s*H where
/// H = { g^j | j in 0..(N-1) } and N is a power of 2.
///
///   1. p(z) = M(z) * \sum_j p(s*g^j) / (dj * (z - s*g^j))
///   2. dj = \prod_{k != j} (s*g^j - s*g^k)
///         = s^{N-1} * \prod_{k != j} (g^j - g^k)
///         = s^{N-1} * N/g^j
///   3. M(z) = \prod_j (z - s*g^j) = z^N - s^N
///
/// therefore, we have
///   p(z) = (M(z)/(s^{N-1}*N)) * \sum_j p(s*g^j) * (g^j / (z-s*g^j))
///        = c * dot_product([p(s*g^j)/(z-s*g^j)],  [g^j])
///   where c = M(z)/(s^{N-1}*N)
///
///
/// we can generalize this formula to get the random linear combination of
/// evaluations of m_i(x) at z. where `m` is a matrix and `m_i(x)` is the i-th
/// column of `m`.
///
/// therefore, we have
///     m(z) = \sum alpha^i * m_i(z)
///          = c * \sum_j [\sum alpha^i * m_i(s*g^j)/(z-s*g^j)] * g^j
///          = c * dot_product([m_rlc], [g^j])
///
/// the evaluation of m(z) is done in three steps:
/// 1. let m_rlc be a vector of size N such that m_rlc[j] = \sum alpha^i *
/// m_i(s*g^j) / (z-s*g^j)
/// 2. let g_powers be a vector of size N such that g_powers[j] = g^j
/// 3. m(z) = c * dot_product(m_rlc, g_powers).
///
/// the above method requires that `m`, `z_diff_invs` and `g_powers` have same
/// order.
//////////////////////////////////////////////////////////////////////////////////////////////

// 1. scale_vec(matrix.rows[i], g_powers[i] * diff_invs[i])
// 2. sum(matrix.rows[i]...matrix.rows[i+TILE_WIDTH])
// this kernel returns the evaluation of each polynomial m_i(x) at z.

static const uint32_t REDUCTION_THREADS_PER_BLOCK = 256;

/// matrix could have bigger height than domain size
/// in this case we shoud do bit_rev twice (first time for domain height, second time for matrix
/// height)
__global__ void matrix_scale_rows_then_reduce(
    FpExt *o_matrix,
    Fp *matrix,
    FpExt *diff_invs_dot_g_powers,
    uint32_t width,
    uint32_t matrix_height,
    uint32_t domain_height
) {
    uint32_t tid = threadIdx.x;
    uint64_t row_index = blockIdx.x * blockDim.x + tid;
    uint64_t col_index = blockIdx.y;

    __shared__ FpExt s_mem[REDUCTION_THREADS_PER_BLOCK];
    s_mem[threadIdx.x] = FpExt(0);

    FpExt sum = FpExt(0);
    uint64_t row_stride = gridDim.x * blockDim.x;
    for (uint32_t i = row_index; i < domain_height; i += row_stride) {
        if (i < domain_height) {
            uint32_t br_i = (domain_height == matrix_height)
                                ? i
                                : bit_rev(bit_rev(i, domain_height), matrix_height);
            sum += FpExt(matrix[col_index * matrix_height + br_i]) * diff_invs_dot_g_powers[i];
        }
    }
    s_mem[tid] = sum;
    __syncthreads();

    for (uint32_t s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            s_mem[tid] += s_mem[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        uint64_t out_height_size = gridDim.x;
        uint64_t out_height_index = blockIdx.x;
        o_matrix[col_index * out_height_size + out_height_index] = s_mem[0];
    }
}

__global__ void matrix_reduce(
    FpExt *o_matrix,
    FpExt *i_matrix,
    uint32_t width,
    uint32_t current_height, // the involved height of the matrix, in current round
    uint32_t buffer_height   // the original height of the matrix
) {
    uint32_t tid = threadIdx.x;
    uint64_t row_index = blockIdx.x * blockDim.x + tid;
    uint64_t col_index = blockIdx.y;

    __shared__ FpExt s_mem[REDUCTION_THREADS_PER_BLOCK];
    s_mem[threadIdx.x] = FpExt(0);

    FpExt sum = FpExt(0);
    uint64_t row_stride = gridDim.x * blockDim.x;
    for (uint64_t i = row_index; i < current_height; i += row_stride) {
        if (i < current_height) {
            sum += i_matrix[col_index * buffer_height + i];
        }
    }
    s_mem[tid] = sum;
    __syncthreads();

    for (uint32_t s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            s_mem[tid] += s_mem[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        uint64_t out_height_index = blockIdx.x;
        o_matrix[col_index * buffer_height + out_height_index] = s_mem[0];
    }
}

__global__ void matrix_get_first_column(
    FpExt *o_col,
    FpExt *i_matrix,
    uint32_t width,
    uint32_t height
) {
    uint32_t gidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (gidx < width) {
        o_col[gidx] = i_matrix[gidx * height];
    }
}

// the quotient polynomial of [m(z) - \sum alpha^i * m_i(x)] divide by (z - x)is
// given by its evaluations at coset domain s*H.
//
// qm(x) = \sum alpha^i * \sum_j (m_i(z)-m_i(x)) / (z-x)
//       = 1/(z-x) * \sum alpha^i * (m_i(z) - m_i(x))
//       = 1/(z-x) * [\sum alpha^i * m_i(z) - \sum alpha^i * m_i(z)]
//       = m(z) / (z-x) - [\sum alpha^i * m_i(x)/(z-x)]
//
// key observation: m_rlc[j] is `\sum alpha^i * m_i(s*g^j)/(z-s*g^j)`.
// therefore, we can reused the result of kernel `matrix_interpolate_coset`.
//
// qm(x) = m(z) / (z-x) - m_rlc(x)
//
// this kernel requires that `acc`, `z_diff_invs` and `m_rlc` have same order.
// for each row, this kernel computes m_rlc[j] = \sum alpha^i * m_i(s*g^j)
__global__ void reduce_matrix_quotient_acc(
    FpExt *quotient_acc,
    Fp *matrix,
    FpExt *z_diff_invs,
    const FpExt *matrix_eval,
    FpExt *d_alphas,
    FpExt *d_alphas_offset,
    uint32_t width,
    uint32_t height,
    bool is_first
) {
    uint32_t row_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (row_idx >= height) {
        return;
    }

    FpExt accum = {0, 0, 0, 0};

    // matrix has a natural order, but all other arrays are bit_reversed
    // so we need to bit_rev when read
    uint32_t br_row_idx = bit_rev(row_idx, height);
    for (uint32_t col_idx = 0; col_idx < width; col_idx++) {
        if (col_idx < width) {
            accum += d_alphas[col_idx] * matrix[col_idx * height + br_row_idx];
        }
    }

    FpExt mz = *matrix_eval;
    FpExt alpha_offset = *d_alphas_offset; // alpha^matrix_offset
    FpExt quotient = alpha_offset * z_diff_invs[row_idx] * (mz - accum);
    if (is_first) {
        quotient_acc[row_idx] = quotient;
    } else {
        quotient_acc[row_idx] += quotient;
    }
}

__global__ void cukernel_split_ext_poly_to_base_col_major_matrix(
    Fp *d_matrix,
    FpExt *d_poly,
    uint64_t poly_len,
    uint32_t matrix_height
) {
    uint32_t row_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (row_idx >= matrix_height) {
        return;
    }

    // d_poly is bit_reversed, so we need to bit_rev when write to keep the natural order
    uint32_t br_row_idx = bit_rev(row_idx, matrix_height);
    uint32_t col_num = (poly_len / matrix_height); // SPLIT_FACTOR = 2
    for (uint32_t col_idx = 0; col_idx < col_num; col_idx++) {
        FpExt ext_val = d_poly[row_idx * col_num + col_idx];
        d_matrix[(col_idx * 4 + 0) * matrix_height + br_row_idx] = ext_val.elems[0];
        d_matrix[(col_idx * 4 + 1) * matrix_height + br_row_idx] = ext_val.elems[1];
        d_matrix[(col_idx * 4 + 2) * matrix_height + br_row_idx] = ext_val.elems[2];
        d_matrix[(col_idx * 4 + 3) * matrix_height + br_row_idx] = ext_val.elems[3];
    }
}

// END OF gpu-backend/src/cuda/kernels/fri.cu

static const size_t FRI_MAX_THREADS = 256;
int get_num_sms() {
    static int multiprocessorCount = []() {
        hipDeviceProp_t prop;
        int device;
        hipGetDevice(&device);
        hipGetDeviceProperties(&prop, device);
        return prop.multiProcessorCount;
    }();
    return multiprocessorCount;
}

extern "C" int _compute_diffs(FpExt *diffs, FpExt *d_z, Fp *d_domain, uint32_t log_max_height) {
    auto block = FRI_MAX_THREADS;
    auto grid = get_num_sms() * 2;
    compute_diffs<<<grid, block>>>(diffs, d_z, d_domain, log_max_height);
    return hipGetLastError();
}

extern "C" int _fri_bit_reverse(FpExt *diffs, uint32_t log_max_height) {
    auto block = FRI_MAX_THREADS;
    auto grid = get_num_sms() * 2;
    fri_bit_reverse<<<grid, block>>>(diffs, log_max_height);
    return hipGetLastError();
}

extern "C" int _batch_invert(FpExt *diffs, uint32_t log_max_height, uint32_t invert_task_num) {
    auto [grid, block] = kernel_launch_params(invert_task_num, FRI_MAX_THREADS);
    batch_invert<<<grid, block>>>(diffs, log_max_height);
    return hipGetLastError();
}

extern "C" int _powers(Fp *data, Fp *g, uint32_t N) {
    auto block = FRI_MAX_THREADS;
    auto grid = get_num_sms() * 2;
    powers<<<grid, block>>>(data, g, N);
    return hipGetLastError();
}

extern "C" int _powers_ext(FpExt *data, FpExt *g, uint32_t N) {
    auto block = FRI_MAX_THREADS;
    auto grid = get_num_sms() * 2;
    powers_ext<<<grid, block>>>(data, g, N);
    return hipGetLastError();
}

extern "C" int _precompute_diff_powers(
    FpExt *d_output,
    const FpExt *diff_invs,
    const Fp *powers,
    uint32_t N
) {
    auto [grid, block] = kernel_launch_params(N, FRI_MAX_THREADS);
    precompute_diff_powers<<<grid, block>>>(d_output, diff_invs, powers, N);
    return hipGetLastError();
}

extern "C" int _matrix_scale_rows_then_reduce(
    FpExt *o_matrix,
    Fp *matrix,
    FpExt *diff_invs_dot_g_powers,
    uint32_t width,
    uint32_t matrix_height,
    uint32_t domain_height,
    uint32_t reduce_matrix_height
) {
    dim3 grid = dim3(reduce_matrix_height, width);
    auto block = REDUCTION_THREADS_PER_BLOCK;
    matrix_scale_rows_then_reduce<<<grid, block>>>(
        o_matrix, matrix, diff_invs_dot_g_powers, width, matrix_height, domain_height
    );
    return hipGetLastError();
}

extern "C" int _matrix_reduce(
    FpExt *o_matrix,
    FpExt *i_matrix,
    uint32_t width,
    uint32_t current_height,
    uint32_t buffer_height,
    uint32_t next_round_height
) {
    dim3 grid = dim3(next_round_height, width);
    auto block = REDUCTION_THREADS_PER_BLOCK;
    matrix_reduce<<<grid, block>>>(o_matrix, i_matrix, width, current_height, buffer_height);
    return hipGetLastError();
}

extern "C" int _matrix_get_first_column(
    FpExt *o_col,
    FpExt *i_matrix,
    uint32_t width,
    uint32_t height
) {
    auto [grid, block] = kernel_launch_params(width, FRI_MAX_THREADS);
    matrix_get_first_column<<<grid, block>>>(o_col, i_matrix, width, height);
    return hipGetLastError();
}

extern "C" int _reduce_matrix_quotient_acc(
    FpExt *quotient_acc,
    Fp *matrix,
    FpExt *z_diff_invs,
    const FpExt *matrix_eval,
    FpExt *d_alphas,
    FpExt *d_alphas_offset,
    uint32_t width,
    uint32_t height,
    bool is_first
) {
    auto [grid, block] = kernel_launch_params(height, TILE_WIDTH);
    reduce_matrix_quotient_acc<<<grid, block>>>(
        quotient_acc,
        matrix,
        z_diff_invs,
        matrix_eval,
        d_alphas,
        d_alphas_offset,
        width,
        height,
        is_first
    );
    return hipGetLastError();
}

extern "C" int _cukernel_split_ext_poly_to_base_col_major_matrix(
    Fp *d_matrix,
    FpExt *d_poly,
    uint64_t poly_len,
    uint32_t matrix_height
) {
    auto [grid, block] = kernel_launch_params(matrix_height, FRI_MAX_THREADS);
    cukernel_split_ext_poly_to_base_col_major_matrix<<<grid, block>>>(
        d_matrix, d_poly, poly_len, matrix_height
    );
    return hipGetLastError();
}

extern "C" int _cukernel_fri_fold(
    FpExt *result,
    FpExt *folded,
    const FpExt *fri_input,
    FpExt *d_constants,
    Fp *g_invs,
    uint64_t N
) {
    auto [grid, block] = kernel_launch_params(N, FRI_MAX_THREADS);
    cukernel_fri_fold<<<grid, block>>>(result, folded, fri_input, d_constants, g_invs, N);
    return hipGetLastError();
}