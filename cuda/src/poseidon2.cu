#include "hip/hip_runtime.h"
// FROM https://github.com/scroll-tech/plonky3-gpu/blob/openvm-v2/gpu-backend/src/cuda/kernels/poseidon2.cu

// Copyright *
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "poseidon2.cuh"
#include "launcher.cuh"

// all matrices are on natural order, so we need to bit_rev row_idx when write
__global__ void poseidon2_rows_p3_multi_kernel(
    Fp *out,
    const uint64_t
        *matrices_ptr, // matrices[0] is the first matrix, matrices[1] is the second matrix, etc.
    const uint64_t *matrices_col,
    const uint64_t *matrices_row,
    uint64_t row_size,
    uint64_t matrix_num
) {
    uint32_t gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid >= row_size) {
        return;
    }

    uint used = 0;
    Fp cells[CELLS];
    for (int i = 0; i < CELLS; i++) {
        cells[i] = Fp(0);
    }

    for (uint m = 0; m < matrix_num; m++) {
        uint64_t col_size = matrices_col[m];
        Fp *matrix = (Fp *)(matrices_ptr[m]);
        for (uint i = 0; i < col_size; i++) {
            cells[used++] = matrix[i * row_size + gid];
            if (used == CELLS_RATE) {
                poseidon2::poseidon2_mix(cells);
                used = 0;
            }
        }
    }

    if (used != 0 || row_size == 0) {
        poseidon2::poseidon2_mix(cells);
    }

    gid = __brev(gid) >> (__clz(row_size) + 1);
    for (uint i = 0; i < CELLS_OUT; i++) {
        out[CELLS_OUT * gid + i] = cells[i];
    }
}

__global__ void poseidon2_compress_kernel(
    Fp *output,
    const Fp *input,
    uint32_t output_size,
    bool is_inject
) {
    uint32_t gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid >= output_size) {
        return;
    }

    Fp cells[CELLS];
    for (size_t i = 0; i < CELLS_OUT; i++) {
        cells[i] = input[(2 * gid + 0) * CELLS_OUT + i];
        cells[i + CELLS_OUT] = input[(2 * gid + 1) * CELLS_OUT + i];
    }

    poseidon2::poseidon2_mix(cells);
    if (is_inject) {
        // hash_pair(&res, &cur)
        for (uint i = 0; i < CELLS_OUT; i++) {
            cells[i + CELLS_OUT] = output[gid * CELLS_OUT + i];
        }
        poseidon2::poseidon2_mix(cells);
    }

    for (uint i = 0; i < CELLS_OUT; i++) {
        output[gid * CELLS_OUT + i] = cells[i];
    }
}

__global__ void babybear_encode_mont_form_kernel(Fp *inout, uint32_t size) {
    uint32_t gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid >= size) {
        return;
    }

    for (uint i = 0; i < CELLS_OUT; i++) {
        inout[gid * CELLS_OUT + i] = Fp(inout[gid * CELLS_OUT + i].get()); // encode
    }
}

/*
query[0][0,...layers-1]
query[1][0,...layers-1]
...
query[k][0,...layers-1]
*/
__global__ void cukernel_query_digest_layers(
    Fp *d_digest_matrix,          // Fp*, also Digest: CELLS_OUT=8 Fp elements
    const uint64_t *d_layers_ptr, // array of Digest layers
    uint64_t *d_indices,          // uint64_t*, indices to query, size = num_query * num_layer
    uint64_t num_query,           // e.g. 100
    uint64_t num_layer
) // e.g. 23
{
    const uint32_t ELEM_PER_DIGEST = CELLS_OUT; // 8 * Fp
    uint64_t gidx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t layer_idx = gidx / ELEM_PER_DIGEST;
    uint64_t elem_offset = gidx % ELEM_PER_DIGEST; // thread group: [0,..7]
    uint64_t query_idx = blockIdx.y;               // [0, num_query -1]
    if (layer_idx >= num_layer) {                  // [0, layers - 1]
        return;
    }

    Fp *d_layer = (Fp *)d_layers_ptr[layer_idx];
    uint64_t digest_offset = d_indices[query_idx * num_layer + layer_idx];
    Fp digest_elem = d_layer[digest_offset * ELEM_PER_DIGEST + elem_offset];
    // now each thread get 1/ELEM_PER_DIGEST of the digest

    uint64_t output_query_offset = query_idx * num_layer * ELEM_PER_DIGEST;
    uint64_t output_layer_offset = layer_idx * ELEM_PER_DIGEST + elem_offset;
    d_digest_matrix[output_query_offset + output_layer_offset] = digest_elem;
}

// END OF FILE gpu-backend/src/cuda/kernels/poseidon2.cu

extern "C" int _poseidon2_rows_p3_multi(
    Fp *out,
    const uint64_t *matrices_ptr,
    const uint64_t *matrices_col,
    const uint64_t *matrices_row,
    const uint64_t row_size,
    uint64_t matrix_num
) {
    auto [grid, block] = kernel_launch_params(row_size);
    poseidon2_rows_p3_multi_kernel<<<grid, block>>>(
        out, matrices_ptr, matrices_col, matrices_row, row_size, matrix_num
    );
    return hipGetLastError();
}

extern "C" int _poseidon2_compress(
    Fp *output,
    const Fp *input,
    uint32_t output_size,
    bool is_inject
) {
    auto [grid, block] = kernel_launch_params(output_size);
    poseidon2_compress_kernel<<<grid, block>>>(output, input, output_size, is_inject);
    return hipGetLastError();
}

extern "C" int _babybear_encode_mont_form(Fp *inout, uint32_t size) {
    auto [grid, block] = kernel_launch_params(size);
    babybear_encode_mont_form_kernel<<<grid, block>>>(inout, size);
    return hipGetLastError();
}

static const size_t QUERY_DIGEST_THREADS = 128;
static const size_t DIGEST_WIDTH = 8;

extern "C" int _query_digest_layers(
    Fp *d_digest_matrix,
    const uint64_t *d_layers_ptr,
    uint64_t *d_indices,
    uint64_t num_query,
    uint64_t num_layer
) {
    auto block = QUERY_DIGEST_THREADS;
    dim3 grid = dim3(div_ceil(num_layer * DIGEST_WIDTH, block), num_query);
    cukernel_query_digest_layers<<<grid, block>>>(
        d_digest_matrix, d_layers_ptr, d_indices, num_query, num_layer
    );
    return hipGetLastError();
}